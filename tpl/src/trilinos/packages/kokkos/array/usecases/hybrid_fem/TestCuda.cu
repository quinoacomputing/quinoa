
#include <TestBoxMeshFixture.hpp>
#include <Implicit.hpp>
#include <NonLinear.hpp>

#include <KokkosArray_Cuda.hpp>
#include <KokkosArray_Host.hpp>

#include <KokkosArray_Cuda_macros.hpp>
#include <ParallelDataMap_macros.hpp>
#include <TestBoxMeshFixture_macros.hpp>
#include <SparseLinearSystem_macros.hpp>
#include <SparseLinearSystemFill_macros.hpp>
#include <Implicit_macros.hpp>
#include <NonLinear_macros.hpp>
#include <KokkosArray_Clear_macros.hpp>

#include <SparseLinearSystem_Cuda.hpp>

//----------------------------------------------------------------------------

void test_cuda_query( comm::Machine machine )
{
  const size_t comm_rank = comm::rank( machine );
  const size_t dev_count = KokkosArray::Cuda::detect_device_count();
  std::cout << "P" << comm_rank
            << ": Cuda device_count = "
            << KokkosArray::Cuda::detect_device_count()
            << std::endl ;
}

//----------------------------------------------------------------------------

void test_cuda_fixture( comm::Machine machine ,
                        size_t nx , size_t ny , size_t nz )
{
  const size_t comm_rank = comm::rank( machine );
  const size_t comm_size = comm::size( machine );
  const size_t dev_count = KokkosArray::Cuda::detect_device_count();
  const size_t dev_rank =
    dev_count && dev_count <= comm_size ? comm_rank % dev_count : 0 ;

  KokkosArray::Cuda::SelectDevice select_device( dev_rank );
  KokkosArray::Cuda::initialize( select_device );
  test_box_fixture<KokkosArray::Cuda>( machine , nx , ny , nz );
  KokkosArray::Cuda::finalize();
}

//----------------------------------------------------------------------------

void test_cuda_implicit( comm::Machine machine , 
                         size_t node_count_begin ,
                         size_t node_count_end ,
                         size_t count_run )
{
  const size_t comm_rank = comm::rank( machine );
  const size_t comm_size = comm::size( machine );
  const size_t dev_count = KokkosArray::Cuda::detect_device_count();
  const size_t dev_rank =
    dev_count && dev_count <= comm_size ? comm_rank % dev_count : 0 ;

  KokkosArray::Cuda::SelectDevice select_device( dev_rank );
  KokkosArray::Cuda::initialize( select_device );
  HybridFEM::Implicit::driver<double,KokkosArray::Cuda>( "Cuda" , machine , node_count_begin , node_count_end , count_run );
  KokkosArray::Cuda::finalize();
}

//----------------------------------------------------------------------------

void test_cuda_nonlinear( comm::Machine machine , 
                          size_t node_count_begin ,
                          size_t node_count_end ,
                          size_t count_run )
{
  const size_t comm_rank = comm::rank( machine );
  const size_t comm_size = comm::size( machine );
  const size_t dev_count = KokkosArray::Cuda::detect_device_count();
  const size_t dev_rank =
    dev_count && dev_count <= comm_size ? comm_rank % dev_count : 0 ;

  KokkosArray::Cuda::SelectDevice select_device( dev_rank );
  KokkosArray::Cuda::initialize( select_device );
  HybridFEM::NonLinear::driver<double,KokkosArray::Cuda>( "Cuda" , machine , node_count_begin , node_count_end , count_run );
  KokkosArray::Cuda::finalize();
}

//----------------------------------------------------------------------------

