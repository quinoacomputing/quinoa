#include "hip/hip_runtime.h"
// Test for boxmuller.h on CUDA
#include <Random123/philox.h>
#include <Random123/threefry.h>
#include "util.h"   // for timer()
#include "util_cuda.h"	// for cuda_init, CHECKCALL
#include "boxmuller.hpp"

typedef r123::Philox4x32 CBRNGF;
typedef r123::Threefry2x64 CBRNGD;
int debug = 0;
const char *progname = "time_boxmuller_cuda";

// Sometimes warnings are A LOT more trouble than they're worth.
// if we just write u[6], we get warnings
// so we write u[(csize>n)?6:0].
#define UGLY(n) (csize>n)?n:0

// The timedloop kernel sums N randoms per thread for timing and
// records that sum in out[tid] (mainly to ensure that
// the random generation process does not get optimized away)
template <typename CBRNG, typename F, typename F2>
__global__ void timedloop(F *out, typename CBRNG::ukey_type k, size_t N){
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t klast = sizeof(k)/sizeof(k[0]) - 1;
    R123_ASSERT(k[klast] == 0); // uses last element of key to
    k[klast] = tid;		// ensure unique key per thread
    F sum = 0.f;
    typename CBRNG::ctr_type ctr = {};
    const size_t csize = sizeof(ctr)/sizeof(ctr[0]);
    CBRNG rng;

    for(size_t i=0; i<N; i+=csize){
        ctr.incr();
        typename CBRNG::ctr_type u = rng(ctr, k);
	F2 f2;
	// Using a loop instead of the Duff device here costs 10%,
	// at least in CUDA4.2 circa Jan 2013 on a Tesla C2050!
	switch(csize) {
	case 8: f2 = r123::boxmuller(u[UGLY(6)], u[UGLY(7)]); sum += f2.x + f2.y;
		f2 = r123::boxmuller(u[UGLY(4)], u[UGLY(5)]); sum += f2.x + f2.y;
	case 4: f2 = r123::boxmuller(u[UGLY(2)], u[UGLY(3)]); sum += f2.x + f2.y;
	case 2: f2 = r123::boxmuller(u[0], u[1]); sum += f2.x + f2.y;
	        break;
	default:
	        R123_ASSERT(0);
	}
    }
    out[tid] = sum;
}

// The dumploop kernel records all the normal randoms individually in out,
// so it produces N randoms per thread.  Each thread records
// its randoms in tid, NTHREADS+tid, NTHREAD*2+tid, ..., NTHREADS*(N-1)+tid
// which hopefully results in nicely coalesced writes from each warp.
template <typename CBRNG, typename F, typename F2>
__global__ void dumploop(F *out, typename CBRNG::ukey_type k, size_t N){
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t klast = sizeof(k)/sizeof(k[0]) - 1;
    R123_ASSERT(k[klast] == 0); // uses last element of key to
    k[klast] = tid;		// ensure unique key per thread
    typename CBRNG::ctr_type ctr = {};
    const size_t csize = sizeof(ctr)/sizeof(ctr[0]);
    CBRNG rng;

    for(size_t i=0; i<N;){
        ctr.incr();
        typename CBRNG::ctr_type u = rng(ctr, k);
	F2 f2;
	// Using a loop instead of the Duff device here costs 10%,
	// at least in CUDA4.2 circa Jan 2013 on a Tesla C2050!
	switch(csize) {
	case 8: f2 = r123::boxmuller(u[UGLY(6)], u[UGLY(7)]);
		out[blockDim.x*gridDim.x*i + tid] = f2.x;
		i++;
		out[blockDim.x*gridDim.x*i + tid] = f2.y;
		i++;
		f2 = r123::boxmuller(u[UGLY(4)], u[UGLY(5)]);
		out[blockDim.x*gridDim.x*i + tid] = f2.x;
		i++;
		out[blockDim.x*gridDim.x*i + tid] = f2.y;
		i++;
	case 4: f2 = r123::boxmuller(u[UGLY(2)], u[UGLY(3)]);
#undef UGLY
		out[blockDim.x*gridDim.x*i + tid] = f2.x;
		i++;
		out[blockDim.x*gridDim.x*i + tid] = f2.y;
		i++;
	case 2: f2 = r123::boxmuller(u[0], u[1]);
		out[blockDim.x*gridDim.x*i + tid] = f2.x;
		i++;
		out[blockDim.x*gridDim.x*i + tid] = f2.y;
		i++;
		break;
	default:
		asm("trap;");
	}
    }
}

template <typename CBRNG, typename F, typename F2>
void timedcall(const char *tname, const char *out_fname, CUDAInfo *infop, typename CBRNG::ukey_type k, size_t N) {
    double cur_time, dt;
    const int nthreads = infop->blocks_per_grid*infop->threads_per_block;
    const size_t nrand = out_fname ? N * nthreads : nthreads;
    const size_t out_size = nrand*sizeof(F);
    F *d_out, *h_out = (F *) malloc(out_size);
    CHECKNOTZERO(h_out);
    CHECKCALL(hipMalloc(&d_out, out_size));
    (void) timer(&cur_time);
    if (out_fname)
	dumploop<CBRNG,F,F2> <<<infop->blocks_per_grid, infop->threads_per_block>>> (d_out, k, N);
    else
	timedloop<CBRNG,F,F2> <<<infop->blocks_per_grid, infop->threads_per_block>>> (d_out, k, N);
    CHECKCALL(hipDeviceSynchronize());
    CHECKCALL(hipMemcpy(h_out, d_out, out_size, hipMemcpyDeviceToHost));
    dt = timer(&cur_time);
    printf("%s %zd in %g sec: %gM/sec\n", tname, N*nthreads, dt, N*nthreads*1.e-6/dt);
    if (out_fname) {
	char *fname = (char *) malloc(strlen(out_fname) + strlen(tname) + 2);
	CHECKNOTZERO(fname);
	sprintf(fname, "%s-%s", out_fname, tname);
	FILE *fp = fopen(fname, "w");
	CHECKNOTZERO(fp);
	for (size_t i = 0; i < nrand; i++){
	    fprintf(fp, "%g\n", h_out[i]);
	}
	fclose(fp);
	free(fname);
    } else {
	int nwoops = 0;
	printf("%s h_out[0] = %g\n", tname, h_out[0]);
	for (size_t i = 0; i < nrand; i++){
	    if(h_out[i] == 0.f){
		if(nwoops++<10)
		    printf("Woops %s h_out[%zd] = %g\n", tname, i, h_out[i]);

	    }
	}
	if(nwoops>10){
	    printf("Woops %s %d times\n", tname, nwoops);
	}
    }
    CHECKCALL(hipFree(d_out));
    free(h_out);
}

const size_t DEF_N = 200000;

int main(int argc, char **argv){
    CBRNGF::ukey_type keyf = {};
    CBRNGD::ukey_type keyd = {};
    size_t Ntry = DEF_N;
    char *cp = getenv("R123_DEBUG");
    if (cp)
	debug = atoi(cp);
    if ((cp = getenv("BOXMULLER_DUMPFILE")) != NULL) {
	Ntry = 8;
    } else {
	Ntry = DEF_N;
    }
    if(argc>1) {
	if (argv[1][0] == '-') {
	    fprintf(stderr, "Usage: %s [iterations_per_thread [key0 [key1]]]\n", argv[0]);
	    exit(1);
	}
        Ntry = atol(argv[1]);
    }
    // XXX cannot use keyf.size in host code, only in device code
    for (int i = 0; i < (int)(sizeof(keyf)/sizeof(keyf[0])-1) && 2+i < argc; i++) {
	keyf.v[i] = atol(argv[2+i]);
    }
    for (int i = 0; i < (int)(sizeof(keyd)/sizeof(keyd[0])-1) && 2+i < argc; i++) {
	keyd.v[i] = atol(argv[2+i]);
    }
    CUDAInfo *infop = cuda_init(getenv("R123_CUDA_DEVICE"));
    timedcall<CBRNGF,float,r123::float2>("float", cp, infop, keyf, Ntry);
    timedcall<CBRNGD,double,r123::double2>("double",cp, infop, keyd, Ntry);
    cuda_done(infop);
    return 0;
}
    
