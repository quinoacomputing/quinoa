
#include <iostream>
#include <hip/hip_runtime.h>


#include <vector>

__global__
void cuda_kernel_init(float* x, int n, float init_value)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n) {
    x[i] = init_value;
  }
}

float* cuda_alloc_float(size_t n, const float& init_value)
{
  float *device_mem = NULL;
  hipError_t err = hipSuccess;

  size_t num_bytes = n*sizeof(float);

  err = hipMalloc((void**)&device_mem, num_bytes);
  if (err != hipSuccess) {
    std::cerr<<"cudaMalloc(device_mem) ERROR: "<<hipGetErrorString(err)<<std::endl;
    return NULL;
  }

  //the following two numbers are magic that I got from a cuda-sample program.
  //I need to understand and possibly parameterize these numbers.

  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

  cuda_kernel_init<<<blocksPerGrid, threadsPerBlock>>>(device_mem, n, init_value);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr<<"cuda_kernel_init ERROR: "<<hipGetErrorString(err)<<std::endl;
    return NULL;
  }

  return device_mem;
}

void copy_cuda_memory_to_host(size_t n, const float* device_mem, std::vector<float>& host_vec)
{
  host_vec.resize(n);

  size_t num_bytes = n*sizeof(float);

  hipError_t err = hipMemcpy((void*)&host_vec[0], (void*)device_mem, num_bytes, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    std::cerr<<"cudaMemcpy(host_vec, device_mem) ERROR: "<<hipGetErrorString(err)<<std::endl;
  }
}

